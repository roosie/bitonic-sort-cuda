#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

hipError_t bitonicWithCuda(int *a, unsigned int size);

__global__ void bitonicKernel(int* a, int nodes, int sets, int core, int index, int allDown, int parentIndex)
{
	int i = threadIdx.x;
	int leftIndex = ((nodes / sets) * (i / (core / sets))) + (i % (core / sets));
	int rightIndex = leftIndex ^ (1 << (index));
	int direction = ((i / (core / (nodes / 2 / (0 ^ (1 << (parentIndex))))))) % 2;

	if (allDown == 1) {
		direction = 0;
	}

	//printf("pre %d: %d;  %d: %d;  alldown: %d; direction: %d \n", leftIndex, a[leftIndex], rightIndex, a[rightIndex], allDown, direction);

	if (a[leftIndex] > a[rightIndex] && (direction == 0)) {
		int temp = a[leftIndex];
		a[leftIndex] = a[rightIndex];
		a[rightIndex] = temp;
	}
	else if (a[leftIndex] < a[rightIndex] && (direction == 1)) {
		int temp = a[leftIndex];
		a[leftIndex] = a[rightIndex];
		a[rightIndex] = temp;
	}
	//printf("post %d: %d;  %d: %d; direction: %d \n", leftIndex, a[leftIndex], rightIndex, a[rightIndex], direction);
}
__global__ void print(int* a) {
	if (threadIdx.x == 0) {
		for (int i = 0; i < 8; i++) {
			printf("%d ", a[i]);
		}
		printf("\n");
	}
}

int main()
{
    const int arraySize = 32;
    int a[arraySize] = { 1,2,3,4,8,7,6,5,1,2,3,4,8,7,6,5,3,4,2,1,66,77,5,4,3,4,5,6,789,5,4,3 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = bitonicWithCuda(a, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	for (int i = 0; i < arraySize; i++) {
		std::cout << a[i] << " ";
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t bitonicWithCuda(int *a, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each pair of elements.

	//(int *a, int nodes, int sets, int core, int index, int allDown)
	int index = 0;
	int allDown = 0;
	for (int i = 0; 0 ^ (1 << (i)) < size; i++) {
		index = i;
		if (0 ^ (1 << (i)) == size) {
			allDown = 1;
		}
		while (index > -1) {
			bitonicKernel << <1, size / 2 >> > (dev_a, size, size/2/( 0 ^ (1 << (index))), size/2, index, allDown, i);
			index--;

			//print << <1, 1 >> > (dev_a);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				goto Error;
			}
			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}
			
		}
	}





Error:
    hipFree(dev_a);
    
    return cudaStatus;
}
